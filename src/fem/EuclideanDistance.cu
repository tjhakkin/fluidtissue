#include "hip/hip_runtime.h"
#include <cfloat>
#include <iostream>
#include "EuclideanDistance.hpp"


namespace {

static const int THREADS_PER_BLOCK = 512;


//
// Computes the shortest distance from mesh node (p; 3) to the interface (ifp; 3xn).
//
__host__ __device__
float distanceToInterface_( const float* p, const float* ifp, const int n_ifp )
{
    float dmin = FLT_MAX;
    
    for (size_t j=0; j<n_ifp; j++) {
        float dx = ifp[3*j+0] - p[0];
        float dy = ifp[3*j+1] - p[1];
        float dz = ifp[3*j+2] - p[2];

#ifdef __CUDA_ARCH__
        float d = norm3df( dx, dy, dz );
#else
        // Note: std::hypot() would be better, but requires C++17.
        float d = std::sqrt( dx*dx + dy*dy + dz*dz );
#endif
        if (d < dmin)
            dmin = d;
    }
    
    return dmin;
}

}   // END namespace


//
// Kernels begin
//


__global__ 
void distanceToInterface_k( const float* meshp, const int n_meshp, 
                            const float* ifp, int n_ifp, double* phi )
{
    int k = threadIdx.x + blockIdx.x*blockDim.x;
    if (k >= n_meshp)
        return;

    float p[3] = {meshp[3*k], meshp[3*k+1], meshp[3*k+2]};
    double dmin = distanceToInterface_( p, ifp, n_ifp );
    if (phi[k] < 0.0f)
        dmin = -dmin;
    phi[k] = dmin;
}


//
// Kernels end
//


void EuclideanDistance::init( const float* meshp, const double* phi0, const int n,
                              const int nThreads )
{
    m_meshSize = n;
    m_threads = nThreads;

    hipGetDeviceCount( &m_devCount );

    if (m_devCount) {
        int size = 3 * n * sizeof(float);
        hipMalloc( &d_p, size );
        hipMemcpy( d_p, meshp, size, hipMemcpyHostToDevice );

        size = n * sizeof(double);
        hipMalloc( &d_phi, size );
        hipMemcpy( d_phi, phi0, size, hipMemcpyHostToDevice );
    }
    else {
        h_p = meshp;
    }
}



void EuclideanDistance::resetPhi( const float* ifp, const int n_ifp, double* phi_n )
{
    if (m_devCount) {
        int blockSize = THREADS_PER_BLOCK;
        int gridSize = (int)ceil( (float)m_meshSize / blockSize );

        float* d_ifp;
        int size = 3 * n_ifp * sizeof(float);
        hipMalloc( &d_ifp, size );
        hipMemcpy( d_ifp, ifp, size, hipMemcpyHostToDevice );

        size = m_meshSize * sizeof(double);
        hipMemcpy( d_phi, phi_n, size, hipMemcpyHostToDevice );
        distanceToInterface_k<<< gridSize, blockSize >>>( d_p, m_meshSize, 
                                                          d_ifp, n_ifp, d_phi );   
        hipFree( d_ifp );
        hipMemcpy( phi_n, d_phi, size, hipMemcpyDeviceToHost );
    }
    else {
        #pragma omp parallel for num_threads( m_threads )
        for (size_t i=0; i<m_meshSize; i++) {
            float p[3] = {h_p[3*i], h_p[3*i+1], h_p[3*i+2]};
            float dmin = distanceToInterface_( p, ifp, n_ifp );
            if (phi_n[i] < 0.0f)
                dmin = -dmin;
            phi_n[i] = dmin;
        }
    }
}
